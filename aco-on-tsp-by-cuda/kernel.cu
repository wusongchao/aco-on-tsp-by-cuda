#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <cfloat>
#include <string>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <cmath>
#include <random>
#include <map>
#include <algorithm>
#include <fstream>

#define CUDA_CALL(x) {const hipError_t a = (x);if(a!=hipSuccess){printf("\nCUDA Error:%s(err_num=%d)\n",hipGetErrorString(a),a);}}

typedef struct Point {
	int index;
	double x;
	double y;
}Point;

__device__ void warpReduce(int tid, float in, float* data)
{
	int idx = (2 * tid - (tid & 0x1f));
	data[idx] = 0;
	idx += 32;
	float t = data[idx] = in;

	data[idx] = t = t + data[idx - 1];
	data[idx] = t = t + data[idx - 2];
	data[idx] = t = t + data[idx - 4];
	data[idx] = t = t + data[idx - 8];
	data[idx] = t = t + data[idx - 16];
}

constexpr int MAX_POINT_NUM = 256;

__inline__ __device__ void clearTauMatrix(unsigned int tid, float* src)
{
	src[tid] = 0.0f;
}

__inline__ __device__ void extractTSPData(int tid, int firstDimension, int secondDimension, struct Point* points, float* graphMatrix, float* heuristicValue, int cityNum)
{
	if (firstDimension != secondDimension && firstDimension < cityNum && secondDimension < cityNum) {
		struct Point lhs = points[firstDimension];
		struct Point rhs = points[secondDimension];
		float val = sqrtf((lhs.x - rhs.x)*(lhs.x - rhs.x) + (lhs.y - rhs.y)*(lhs.y - rhs.y));
		graphMatrix[tid] = val;
		heuristicValue[tid] = 1 / val;
	}
}

__inline__ __device__ void initializeTauState(int tid, float* Tau, float* deltaTau)
{
	Tau[tid] = 1.0f;
	deltaTau[tid] = 0.0f;
}

__global__ void clearTabu(bool* Tabu)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	Tabu[tid] = true;
}

__global__ void preProcessing(struct Point* points, float* graphMatrix, float* heuristicValue, float* Tau, float* deltaTau, int cityNum)
{
	// to reduce memory request
	int firstDimension = blockIdx.x;
	int secondDimension = threadIdx.x;
	int tid = firstDimension * MAX_POINT_NUM + secondDimension;
	extractTSPData(tid, firstDimension, secondDimension, points, graphMatrix, heuristicValue, cityNum);
	initializeTauState(tid, Tau, deltaTau);
}

// for each block(ant), we have only one "randomEngine"
__inline__ __device__ int generateInitialCityIndex(hiprandState* state, int cityNum)
{
	// since city index in array will begin with 0
	return hiprand(state) % cityNum;
}

__shared__ int path[MAX_POINT_NUM];
__shared__ int current;
__shared__ int startCityIndex;
__shared__ float currentDistance;
__shared__ float probabilities[MAX_POINT_NUM];
__shared__ float denominatorSum;
__shared__ bool allowed[MAX_POINT_NUM];


__inline__ __device__ void calculateProbability(int tid, float* Tau, const float* heuristicValue, int cityNum, float alpha, float beta)
{
	// each thread mapped to a city(start from zero)
	if (allowed[tid] == true) {
		const int offset = MAX_POINT_NUM;
		// must ensure that val not all is 0, so, have to make Tau an initial value instead of 0
		float val = __powf(Tau[current * offset + tid], alpha)*__powf(heuristicValue[current * offset + tid], beta);
		// means possibility from current to tid
		probabilities[tid] = val;
		atomicAdd(&denominatorSum, val);
	}
}

__inline__ __device__ int selectCity(hiprandState* state, int cityNum)
{
	float generatedProbablity = hiprand_uniform(state);
	float sumSelect = 0.0f;
	int selectedIndex = 0;
	for (int i = 0; i < cityNum;i++) {
		if (allowed[i] == true) {
			sumSelect += (probabilities[i] / denominatorSum);
			if (sumSelect >= generatedProbablity) {
				selectedIndex = i;
				break;
			}
		}
	}
	return selectedIndex;
}

__global__ void constructPath(float* Tau, float* deltaTau, const float* graphMatrix, const float* heuristicValue, hiprandState* states, int cityNum, float alpha, float beta, int Q, 
	float* bestLength, int* bestPath)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	const int offset = MAX_POINT_NUM;
	
	if (tid < cityNum) {
		int innerPathSize = 0;
		allowed[tid] = true;
		if (tid == 0) {
			int temp = generateInitialCityIndex(states + bid, cityNum);
			path[innerPathSize] = current = startCityIndex = temp;
			currentDistance = 0.0f;
			allowed[current] = false;
			denominatorSum = 0.0f;
		}
		innerPathSize = 1;
		__syncthreads();

		while (innerPathSize < cityNum) {
			calculateProbability(tid, Tau, heuristicValue, cityNum, alpha, beta);
			__syncthreads();

			if (tid == 0) {
				int next = selectCity(states + bid, cityNum);
				allowed[next] = false;
				currentDistance += graphMatrix[current * offset + next];
				path[innerPathSize] = next;
				current = next;
				denominatorSum = 0;
			}
			innerPathSize++;
			__syncthreads();
		}

		if (tid == 0) {
			currentDistance += graphMatrix[current * offset + startCityIndex];
			float bestLengthForThisAnt = bestLength[bid];
			if (currentDistance < bestLengthForThisAnt) {
				bestLength[bid] = currentDistance;
				int off = bid * offset;
				for (int i = 0; i < cityNum;i++) {
					bestPath[off + i] = path[i];
				}
			}
		}
		else {
			float val = Q / currentDistance;
			atomicAdd(&deltaTau[path[tid - 1] * offset + path[tid]], val);
			atomicAdd(&deltaTau[path[tid] * offset + path[tid - 1]], val);
		}
	}
}

__global__ void updatePheromones(float* Tau, float* deltaTauTotal, double rho, int cityNum)
{
	// blockIdx.x is the first dimension
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float tau = Tau[tid];
	tau = (1 - rho) * tau;
	tau += deltaTauTotal[tid];

	// clear the deltaTau
	deltaTauTotal[tid] = 0.0f;
	Tau[tid] = tau;
}

__global__ void setupCuRandKernel(hiprandState* state)
{
	
    #define THREAD_SEED 1234
	int id = threadIdx.x;
	hiprand_init(THREAD_SEED, id, 0, state + id);
}

__global__ void initializeEachAntsBestLength(float* length)
{
	length[threadIdx.x] = FLT_MAX;
}

// use a sub-blocks to represent the range
// that is, if we have m city left in the unvisited list, we have m sub-blocks
// than use a __ballot(probability in range?) operation to find out which sub-block "win"
//unsigned int __ballot(int x)
//{
//	if (x != 0) {
//		return (1 << (threadIdx.x % 32));
//	}
//	return 0;
//}

int main()
{
	constexpr int antNum = 32;
	constexpr int iterTimes = 600;
	double alpha = 1;//��Ϣ�ص���Ҫ�̶�
	double beta = 3.8;//����ʽ���ӵ���Ҫ�̶�
	double rho = 0.7;//�ӷ�ϵ��
	int left = 2;//��Ϣ������
	int Q = 400;//��Ϣ��������

	std::string filePath;

	std::cin >> filePath;

	std::ifstream in(filePath.c_str());
	if (!in.is_open()) {
		std::cout << "No exist!" << std::endl;
	}

	Point points[MAX_POINT_NUM];
	int index;
	double x, y;
	int cityNum = 0;
	while (in >> index >> x >> y) {
		points[cityNum++] = { index,x,y };
	}
	Point* devicePoints;
	float* deviceGraphMatrix;
	float* deviceHeuristicValue;
	float* deviceTau;
	float* deviceDeltaTau;
	CUDA_CALL(hipMalloc(&devicePoints, sizeof(Point) * MAX_POINT_NUM));
	CUDA_CALL(hipMemcpy(devicePoints, points, sizeof(Point) * MAX_POINT_NUM, hipMemcpyHostToDevice));
	CUDA_CALL(hipMalloc(&deviceGraphMatrix, sizeof(float) * MAX_POINT_NUM * MAX_POINT_NUM));
	CUDA_CALL(hipMalloc(&deviceHeuristicValue, sizeof(float) * MAX_POINT_NUM * MAX_POINT_NUM));
	CUDA_CALL(hipMalloc(&deviceTau, sizeof(float) * MAX_POINT_NUM * MAX_POINT_NUM));
	CUDA_CALL(hipMalloc(&deviceDeltaTau, sizeof(float) * MAX_POINT_NUM * MAX_POINT_NUM));

	preProcessing <<<MAX_POINT_NUM, MAX_POINT_NUM >>>(devicePoints, deviceGraphMatrix, deviceHeuristicValue, deviceTau, deviceDeltaTau, cityNum);

	// init randomEngine
	hiprandState* devStates;
	CUDA_CALL(hipMalloc(&devStates, antNum * sizeof(hiprandState)));
	setupCuRandKernel << <1, antNum >> > (devStates);

	float* deviceBestLength;
	int* deviceBestPath;
	hipMalloc(&deviceBestPath, sizeof(int) * antNum * MAX_POINT_NUM);
	hipMalloc(&deviceBestLength, sizeof(float) * antNum);
	initializeEachAntsBestLength<<<1, antNum>>>(deviceBestLength);

	int iterCounter = 0;

	hipEvent_t startEvent;
	hipEvent_t endEvent;
	float cudaElapsedTime = 0.0f;
	hipEventCreate(&startEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(startEvent, 0);

	while (iterCounter < iterTimes) {
		constructPath<<<antNum, MAX_POINT_NUM>>>(deviceTau,deviceDeltaTau,deviceGraphMatrix,deviceHeuristicValue,devStates,cityNum,alpha,beta,Q,deviceBestLength,deviceBestPath);
		updatePheromones<<<MAX_POINT_NUM, MAX_POINT_NUM>>>(deviceTau,deviceDeltaTau,rho,cityNum);
		iterCounter++;
	}

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&cudaElapsedTime, startEvent, endEvent);

	std::cout << "core kernel time:" << cudaElapsedTime << std::endl;

	float hostBestLength[antNum];
	int hostBestPath[antNum][MAX_POINT_NUM];

	hipMemcpy(hostBestLength, deviceBestLength, sizeof(float) * antNum, hipMemcpyDeviceToHost);
	hipMemcpy(hostBestPath, deviceBestPath, sizeof(int) * antNum * MAX_POINT_NUM, hipMemcpyDeviceToHost);

	float bestLength = hostBestLength[0];
	int pos = 0;
	for (int i = 1;i < antNum;i++) {
		if (hostBestLength[i] < bestLength) {
			bestLength = hostBestLength[i];
			pos = i;
		}
	}

	std::cout << bestLength << std::endl;
	for (int i = 0;i < cityNum;i++) {
		std::cout << hostBestPath[pos][i]+1 << ' ';
	}

	return 0;
}